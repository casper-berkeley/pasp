#define DEF_CUFFT_FUNC

void exec_part_cufft(hipfftComplex *, int, int);
void exec_cufft(hipfftComplex *, int, int);
void check_cufft_plan_error(hipfftResult );
void check_cufft_exec_error(hipfftResult );
void output_cufft_plan_error(char *);
void output_cufft_exec_error(char *);


void exec_part_cufft(hipfftComplex *devData, int fftLen, int fftNum){

	// for using cufft
	hipfftResult  planResult;
	hipfftHandle  plan;
	hipfftResult  fftResult;

	// create cufft plan
	planResult = hipfftPlan1d(&plan, fftLen, HIPFFT_C2C, fftNum/2);
	check_cufft_plan_error(planResult);

	// do fft first
	fftResult = hipfftExecC2C(plan, devData, devData, HIPFFT_FORWARD);
	check_cufft_exec_error(fftResult);
	hipDeviceSynchronize();

	// do fft second
	fftResult = hipfftExecC2C(plan, &devData[fftLen * fftNum / 2], &devData[fftLen * fftNum / 2], HIPFFT_FORWARD);
	check_cufft_exec_error(fftResult);
	hipDeviceSynchronize();

	// destroy cufft plan
	hipfftDestroy(plan);

	return;
}


void exec_cufft(hipfftComplex *devData, int fftLen, int fftNum){

	// for using cufft
	hipfftResult  planResult;
	hipfftHandle  plan;
	hipfftResult  fftResult;

	// create cufft plan
	planResult = hipfftPlan1d(&plan, fftLen, HIPFFT_C2C, fftNum);
	check_cufft_plan_error(planResult);

	// do fft first
	fftResult = hipfftExecC2C(plan, devData, devData, HIPFFT_FORWARD);
	check_cufft_exec_error(fftResult);
	hipDeviceSynchronize();

	// destroy cufft plan
	hipfftDestroy(plan);

	return;
}



void check_cufft_plan_error(hipfftResult result){

	switch(result){
		case HIPFFT_SETUP_FAILED:
			output_cufft_plan_error("CUFFT library failed to initialize");
			break;
		case HIPFFT_INVALID_SIZE:
			output_cufft_plan_error("The nx parameter is not a supported size");
			break;
		case HIPFFT_INVALID_TYPE:
			output_cufft_plan_error("The type parameter is not supported");
			break;
		case HIPFFT_ALLOC_FAILED:
			output_cufft_plan_error("Allocation of GPU resources for the plan failed.");
			break;
		default :
			break;
	}

	return;
}


void check_cufft_exec_error(hipfftResult result){

	switch(result){
		case HIPFFT_SETUP_FAILED :
			output_cufft_exec_error("CUFFT library failed to initialize");
			break;
		case HIPFFT_INVALID_PLAN :
			output_cufft_exec_error("The plan parameter is not a valid handle");
			break;
		case HIPFFT_INVALID_VALUE :
			output_cufft_exec_error("The idata, odata, and/or direction parameter is not valid.");
			break;
		case HIPFFT_EXEC_FAILED :
			output_cufft_exec_error("CUFFT failed to execute the transform on GPU");
			break;
		default:
			break;
	}

	return;
}

void output_cufft_plan_error(char *str){
	fprintf(stderr,"CUFFT PLAN ERROR : %s\n",str);

	return;
}

void output_cufft_exec_error(char *str){
	fprintf(stderr,"CUFFT EXEC ERROR : %s\n",str);

	return;
}

