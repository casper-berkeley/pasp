#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <cutil_inline.h>
#include <hipfft/hipfft.h>
#include "fft_gpu.h"

#define FILENAME_BUFSIZE 200
#define SUM_MAX_THREAD   256
#define SUB_MAX_THREAD   256
#define MAX_THREAD       256
#define MAX_GRID         32*1024
#define LOOP_NUM         1

#include "kernelExec.cu"
#include "fourStepFFT.cu"

#include "output_greg.c"
#include "random.c"




extern "C"
int do_analyze_on_gpu(int signalLength)  {


	// default value of boxcar, this can be change by -boxcar argument
	int  boxcar = 8192;

	// default value of threshold, this can be changed by -threshold argument
	int  threshold = 20;

	// default filename of output file, this can be changed by -output_file argument
	char outputFileName[FILENAME_BUFSIZE] = "report.txt";

	// the memory size and pointer for generated signal data
	unsigned int hostSignalDataMemSize = 0;
	char         *hostSignalData = NULL;
	float        *hostPowerData = NULL;
	hipfftComplex *hostFFTData = NULL;

	// the memory size and pointer for output data
	unsigned int hostPowerDataMemSize = 0;
	unsigned int hostFFTDataMemSize = 0;
	unsigned int hostOutputDataMemSize = 0;
	outputStruct *hostOutputData = NULL;

	// we limit the detected signal points in each boxcar.
	// 'maximumDetectPointInBoxcar' specify its value, this can be changed by -max_detect_point argument
	int maximumDetectPointInBoxcar = 16;

	// output file 
	int   outputCounter = 0;
	int   outputFclosePeriod = 10;
	FILE *outputFilePointer = NULL;





	printf("Init Host memory\n");
	
        // -- Init host memory --  
        hostPowerDataMemSize = sizeof(float) * signalLength;
        cutilSafeCall( hipHostMalloc( (void**)&hostPowerData, hostPowerDataMemSize));
        if(hostPowerData==NULL){
                fprintf(stderr,"Error : hipHostMalloc failed\n");
                exit(-1);
        }

        hostFFTDataMemSize = sizeof(hipfftComplex) * signalLength;
        cutilSafeCall( hipHostMalloc( (void**)&hostFFTData, hostFFTDataMemSize));
        if(hostFFTData==NULL){
                fprintf(stderr,"Error : hipHostMalloc failed\n");
                exit(-1);
        }

        hostSignalDataMemSize = sizeof(char) * signalLength * 2;
        hostOutputDataMemSize = sizeof(outputStruct) * maximumDetectPointInBoxcar * (signalLength / boxcar);

        cutilSafeCall( hipHostMalloc( (void**)&hostSignalData, hostSignalDataMemSize));
        if(hostSignalData==NULL){
                fprintf(stderr,"Error : hipHostMalloc failed\n");
                exit(-1);
        }

        cutilSafeCall( hipHostMalloc( (void**)&hostOutputData, hostOutputDataMemSize));
        if(hostOutputData==NULL){
                fprintf(stderr,"Error : hipHostMalloc failed\n");
                exit(-1);
        }

	
	// -- Device memory pointer
	char         *devSignalData = NULL;
	hipfftComplex *devFFTData    = NULL;
	hipfftComplex *devCuFFTData  = NULL;
	float        *devPowerData  = NULL;
	float        *devAvgRe      = NULL;
	float        *devAvgIm      = NULL;
	float        *devPartSumRe  = NULL;
	float        *devPartSumIm  = NULL;
	outputStruct *devOutputData = NULL;

	// Memory size for device
	unsigned int devSignalDataMemSize = 0;
	unsigned int devFFTDataMemSize    = 0;
	unsigned int devPowerDataMemSize  = 0;
	unsigned int devPartSumMemSize    = 0;
	unsigned int devOutputDataMemSize = 0;


	// Calculate memory size
	devSignalDataMemSize = hostSignalDataMemSize;
	devFFTDataMemSize    = sizeof(hipfftComplex) * signalLength;
	devPowerDataMemSize  = sizeof(float) * signalLength;
	devPartSumMemSize    = sizeof(float) * SUM_MAX_THREAD;
	devOutputDataMemSize    = hostOutputDataMemSize;

	// Allocate device memory
	printf("Allocate device memory\n");
	cutilSafeCall( hipMalloc( (void**) &devSignalData, devSignalDataMemSize) );
	cutilSafeCall( hipMalloc( (void**) &devFFTData,    devFFTDataMemSize) );
	cutilSafeCall( hipMalloc( (void**) &devPowerData,  devPowerDataMemSize) );
	cutilSafeCall( hipMalloc( (void**) &devOutputData, devOutputDataMemSize) );
	cutilSafeCall( hipMalloc( (void**) &devPartSumRe,  devPartSumMemSize) );
	cutilSafeCall( hipMalloc( (void**) &devPartSumIm,  devPartSumMemSize) );
	cutilSafeCall( hipMalloc( (void**) &devAvgRe, sizeof(float) * 1) );
	cutilSafeCall( hipMalloc( (void**) &devAvgIm, sizeof(float) * 1) );

	// the row length and col length of matrix
	int matrixX, matrixY;

	// the value of 'matrixY' must be fixed!!! because this program includes only 16-point fft kernel.
	matrixY = 16;
	matrixX = signalLength / matrixY;

	// Generate signal
	long seed1 = 27;
	long seed2 = 22;

	printf("Generating gaussian numbers\n");
	for(int i=0; i<signalLength; i++){

        	//hostSignalData[2*i] = 0.01 * cosf(i * 8 * 2*3.14159265/(float)signalLength) + gauss(&seed1, 0.0, 1.0);
        	//hostSignalData[2*i+1] = 0.01 * sinf(i * 8 * 2*3.14159265/(float)signalLength) + gauss(&seed2, 0.0, 1.0); 
		hostFFTData[i].x = 0.01 * cosf(i * 800 * 2*3.14159265/(float)signalLength) + gauss(&seed1, 0.0, 1.0);
		hostFFTData[i].y = 0.01 * sinf(i * 800 * 2*3.14159265/(float)signalLength) + gauss(&seed2, 0.0, 1.0);

		//printf("gauss = %f\n",gauss(&seed1, 0.0, 1.0));
	}

	// Initialize output file
	int  result;
	char buf[FILENAME_BUFSIZE];

        result = sprintf(buf,"%s",outputFileName);
        if(result==EOF){
                fprintf(stderr,"Error : sprintf failed in init_output_file()\n");
                return 0;
        }

        //outputFilePointer = fopen(buf,"wb");
        outputFilePointer = fopen(buf,"w");
        if(outputFilePointer==NULL){
                fprintf(stderr,"Error : fopen failed int init_output_file()\n");
                return 0;
        }

	// timer
	unsigned int timer;
	cutCreateTimer(&timer);	

	unsigned int timerA;
	cutCreateTimer(&timerA);	

	unsigned int timerB;
	cutCreateTimer(&timerB);	

	// Cufft
	hipfftHandle plan;
	// CUFFT_SAFE_CALL(hipfftPlan1d(&plan, signalLength, HIPFFT_C2C, 1));
	CUFFT_SAFE_CALL(hipfftPlan1d(&plan, signalLength, HIPFFT_C2R, 1));


	// Main loop
	printf("Main loop\n");
	for(int iter=0; iter<LOOP_NUM; iter++){

		// timer
		cutResetTimer(timer);
		cutResetTimer(timerA);
		cutResetTimer(timerB);

		cutStartTimer(timer);
#if 0
		// CPU -> GPU : move signal data from host to device
		cutilSafeCall( hipMemcpy(devSignalData, hostSignalData, devSignalDataMemSize, hipMemcpyHostToDevice));
		hipDeviceSynchronize();

		// GPU : convert char format signal data to float format
		convert_to_float_exec(devSignalData, devPartSumRe, devPartSumIm, devAvgRe, devAvgIm, devFFTData, signalLength);
#endif
		cutilSafeCall( hipMemcpy( devFFTData, hostFFTData, devFFTDataMemSize, hipMemcpyHostToDevice));

		// GPU : do fft
		cutStartTimer(timerA);
		do_four_step_fft(devFFTData, devPowerData, matrixX, matrixY);
		hipDeviceSynchronize();
		cutStopTimer(timerA);


		// GPU : detect strong power spectrum
		calc_over_threshold_exec(devPowerData, devOutputData, signalLength, boxcar, threshold, maximumDetectPointInBoxcar);


		// GPU -> CPU : copy myfft data from device to host
		cutilSafeCall( hipMemcpy( hostPowerData, devPowerData, devPowerDataMemSize, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();

		// GPU -> CPU : copy detect spectrum data from device to host
                cutilSafeCall( hipMemcpy( hostOutputData, devOutputData, devOutputDataMemSize, hipMemcpyDeviceToHost));
                hipDeviceSynchronize();


		// CPU : output detect power spectrum to file
		//output_spectrum_to_file(outputFilePointer, signalLength, hostSignalData, hostPowerData,  (hipfftReal*)hostcuFFTData);
		//output_spectrum_to_file_float(outputFilePointer, signalLength, hostcuFFTData, hostPowerData, hostcuFFTRData);
		output_spectrum_to_file_float_threshold(outputFilePointer, signalLength, hostFFTData, hostPowerData, hostOutputData, boxcar, maximumDetectPointInBoxcar);

		// timer
		cutStopTimer(timer);
		printf("time = %f  myfft = %f  CuFFT= %f, %d done...\n",cutGetTimerValue(timer), cutGetTimerValue(timerA), cutGetTimerValue(timerB), iter);

	}

	// Terminate output file
	terminate_output_file(outputFilePointer);


	// Free device memory
	cutilSafeCall( hipFree( devSignalData ) );
	cutilSafeCall( hipFree( devFFTData ) );
	cutilSafeCall( hipFree( devPowerData ) );
	cutilSafeCall( hipFree( devPartSumRe ) );
	cutilSafeCall( hipFree( devPartSumIm ) );
	cutilSafeCall( hipFree( devAvgRe ) );
	cutilSafeCall( hipFree( devAvgIm ) );


        // Free signal data memory
        hipHostFree(hostSignalData);
        hostSignalData = NULL;

        // Free output data memory
        hipHostFree(hostOutputData);
        hostOutputData = NULL;

        return 0;
}
