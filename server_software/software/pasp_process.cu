/*
 *  pasp_process.c
 *  
 *
 *  Created by Terry E. Filiba on 2/24/09.
 *
 */

#include <stdlib.h>
#include <stdio.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <string.h>
#include <cutil_inline.h>

#include "pasp_config.h"
#include "pasp_process.h"
#include "fft_library.h"
#include "debug_macros.h"

// size of the fft
static const unsigned int signalLength = 1024*1024;

int main(int argc, char *argv[])
{
    int i=0;
    
    // buffer for the next packet
    char *hostSignalData;
    
    unsigned int hostSignalDataMemSize = sizeof(char) * signalLength * 2;
    cutilSafeCall( hipHostMalloc( (void**)&hostSignalData, hostSignalDataMemSize));
	if(hostSignalData==NULL){
		fprintf(stderr,"Error : hipHostMalloc failed\n");
		exit(-1);
	}
    
    struct sigaction newact;
    
    int numbytes=0;
    int numpackets=0;
    long long totalbytes=0;
    
    // input fifo file info
    int input_fifo;
    char input_file_name[CHANNEL_FILE_NAME_SIZE];
    
    // this should really be a command line opt
    int channelid=8;
    int polid=0;  
    
    float *hostPower;
    
    unsigned int timer;
    cutCreateTimer(&timer);
    
    
    
    //set up the signal handler
    newact.sa_handler = cleanup;
    sigemptyset(&newact.sa_mask);
    newact.sa_flags = 0;
    
    //start listening for Ctrl-C
	sigaction(SIGINT, &newact, NULL);
    
    // open the fifo with complex data for a single channel/pol
    snprintf(input_file_name, CHANNEL_FILE_NAME_SIZE, CHANNEL_FILE_BASE, channelid, polid);
    debug_fprintf(stderr, "Opening fifo %s\n", input_file_name);
    input_fifo = open(input_file_name,O_RDONLY);
    
    initializeFFT(signalLength);
    
    debug_fprintf(stderr, "Waiting for data\n");
    while(run_fifo_read==1)
    {
        // read packet from fifo
        numbytes = read(input_fifo, (void *) &(hostSignalData[i]), hostSignalDataMemSize);
        //fprintf(stderr,"tried to read %d got %d at %x\n", hostSignalDataMemSize, numbytes, (void *) hostSignalData);
        if(numbytes==-1 && run_fifo_read==1)
        {
            perror("Error reading from fifo");
            exit(0);
        }
        
        // process packet
        if(run_fifo_read==1 && numbytes!=0)
        {
            numpackets++;
            totalbytes+=numbytes;
            i+=numbytes;
            if(i>=hostSignalDataMemSize)
            {
                cutResetTimer(timer);
		        cutStartTimer(timer);
                hostPower=callFFT(hostSignalData);
                cutStopTimer(timer);
                
                printf("time = %f done...\n",cutGetTimerValue(timer));
                for(int j=0;j<signalLength;j++)
                {
                    if(hostPower[j] != 0)
                    {
                        fprintf(stdout, "%d\t%f\n", j, hostPower[j]);
                    }
                }
                i=0;
            }

        }
    }
    
    debug_fprintf(stderr, "Received %d packets, %lld bytes\n", numpackets, totalbytes);
    debug_fprintf(stderr, "Closing fifo\n");
    close(input_fifo);
    destroyFFT();
    //free(newdata);
    return 0;
}

static void cleanup(int signal)
{
    debug_fprintf(stderr, "Ctrl-C received... cleaning up\n");
	run_fifo_read = 0;
}

