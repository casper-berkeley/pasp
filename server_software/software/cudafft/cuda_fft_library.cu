#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
#include <cutil_inline.h>


static hipfftHandle plan;
hipfftComplex *hostSignalData;
float *hostPowerData;
hipfftComplex *gpudata;
hipfftComplex *fftgpudata;

int signalLength;

/*******************
REMOVE
*******************/
#define BATCH 1

void initializeFFT(int initializedSignalLength)
{
    signalLength = initializedSignalLength;

    cutilSafeCall( hipHostMalloc( (void**)&hostSignalData, sizeof(hipfftComplex)*signalLength) );
    cutilSafeCall( hipHostMalloc((void**)&hostPowerData, sizeof(float)*signalLength) );
    
    // allocate device memory for the fft
    hipMalloc((void**)&gpudata,sizeof(hipfftComplex)*signalLength);
    hipMalloc((void**)&fftgpudata,sizeof(hipfftComplex)*signalLength);
    
    
    
    hipfftPlan1d(&plan,signalLength,HIPFFT_C2C, BATCH);
}


float * callFFT(char *data)
{
    //convert the data to float
    for(int i=0; i<signalLength; i++)
    {
        hostSignalData[i].x = data[2*i];
        hostSignalData[i].y = data[2*i+1];
    }
    
    // copy data to the gpu
    hipMemcpy(gpudata, hostSignalData, sizeof(hipfftComplex)*signalLength, hipMemcpyHostToDevice);
    
    // run the fft
    
    hipfftExecC2C(plan,gpudata,fftgpudata,HIPFFT_FORWARD);
    hipDeviceSynchronize();
    
    // copy the result back
    hipMemcpy(hostSignalData, fftgpudata, sizeof(hipfftComplex)*signalLength, hipMemcpyDeviceToHost);
    
    for(int i=0;i<signalLength;i++)
    {
        hostPowerData[i] = hostSignalData[i].x*hostSignalData[i].x + hostSignalData[i].y*hostSignalData[i].y;
    }
    
//    for(int i=0; i<signalLength; i++)
//    {
//        fprintf(stderr,"%d %f %f\n", i, hostSignalData[i].x, hostSignalData[i].y);
//    }
    
    return hostPowerData;
}

void destroyFFT()
{
    hipfftDestroy(plan);
    hipHostFree(hostSignalData);
    hipHostFree(hostPowerData);
    hipFree(gpudata);
    hipFree(fftgpudata);
}
