#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

//#include "fft_library.h"
//#include "pasp_config.h"

static hipfftHandle plan;
hipfftComplex *gpudata;
hipfftComplex *fftgpudata;

/*******************
REMOVE
*******************/
#define CHANNEL_BUFFER_SIZE 11
#define NX  16
#define BATCH 10
#define SAMPLES_PER_CHANNEL 5

void initializeFFT()
{
    // allocate device memory for the fft
    hipMalloc((void**)&gpudata,CHANNEL_BUFFER_SIZE*NX*BATCH);
    hipMalloc((void**)&fftgpudata,CHANNEL_BUFFER_SIZE*NX*BATCH);
    
    hipfftPlan1d(&plan,SAMPLES_PER_CHANNEL*NX,HIPFFT_C2C, BATCH);
}


void callFFT(hipfftComplex *data)
{
    //int i;
    // allocate device memory and copy over data
    hipMemcpy(gpudata, data, CHANNEL_BUFFER_SIZE*NX*BATCH, hipMemcpyHostToDevice);
    
    // run the fft
    
    hipfftExecC2C(plan,gpudata,fftgpudata,HIPFFT_FORWARD);
    
    // copy the result back
    hipMemcpy(data, fftgpudata, CHANNEL_BUFFER_SIZE*NX*BATCH, hipMemcpyDeviceToHost);
    
//    for(i=0; i<SAMPLES_PER_CHANNEL*NX*BATCH; i++)
//    {
//        fprintf(stderr,"%d %f %f\n", i, data[i].x, data[i].y);
//    }
    

}

void destroyFFT()
{
    hipfftDestroy(plan);
    hipFree(gpudata);
}
