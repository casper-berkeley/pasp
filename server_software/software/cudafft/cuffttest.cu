#include <stdlib.h>
#include <stdio.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
#include <cutil_inline.h>

//#define NX      256
//#define BATCH   4
#define MAX_NX      16777216
#define MAX_BATCH   4096
#define MAX_DIM     16777216


static hipfftHandle plan;
hipfftComplex *gpudata;
hipfftComplex *fftgpudata;


int main ()
{
    long long i;
    long long nx;
    long long batch;
    unsigned int complete_fft_timer;
    unsigned int copy_to_gpu_timer;
    unsigned int fft_only_timer;
    unsigned int copy_from_gpu_timer;
    
    //hipfftHandle plan;
    hipfftComplex *data;
    hipfftComplex *result;
    //hipfftComplex *gpudata;
    
    hipHostMalloc(&data, sizeof(hipfftComplex)*MAX_DIM);
    hipHostMalloc(&result, sizeof(hipfftComplex)*MAX_DIM);
    
    //fprintf(stderr, "Initializing data... ");
    // generate some random data
    for(i=0; i<MAX_DIM; i++)
    {
        data[i].x=1.0f;
        data[i].y=1.0f;
    }
    //fprintf(stderr, "done\n");
    cutCreateTimer(&complete_fft_timer);
    cutCreateTimer(&copy_to_gpu_timer);
    cutCreateTimer(&fft_only_timer);
    cutCreateTimer(&copy_from_gpu_timer);
    
    //fprintf(stderr, "nx\tbatch\ttime\tcopy_to_gpu\tactual_fft\tcopy_from_gpu\tavg\n");
    for(nx=2; nx<=MAX_NX; nx=nx*2)
    //for(nx=4096; nx<=MAX_NX; nx+=4096)
    {
        for(batch=1;batch<=MAX_BATCH;batch=batch*2)
        {
            if(nx*batch <= MAX_DIM)
            {  
                // allocate device memory for the fft
                CUDA_SAFE_CALL(hipMalloc((void**)&gpudata,sizeof(hipfftComplex)*nx*batch));
                CUDA_SAFE_CALL(hipMalloc((void**)&fftgpudata,sizeof(hipfftComplex)*nx*batch));

                hipfftPlan1d(&plan,nx,HIPFFT_C2C, batch);
                
                cutResetTimer(complete_fft_timer);
                cutStartTimer(complete_fft_timer);
                // run the fft
                // allocate device memory and copy over data
                hipMemcpy(gpudata, data, sizeof(hipfftComplex)*nx*batch, hipMemcpyHostToDevice);
                // run the fft
                hipfftExecC2C(plan,gpudata,fftgpudata,HIPFFT_FORWARD);
                // copy the result back
                hipMemcpy(result, fftgpudata, sizeof(hipfftComplex)*nx*batch, hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                cutStopTimer(complete_fft_timer);
                
                cutResetTimer(copy_to_gpu_timer);
                cutStartTimer(copy_to_gpu_timer);
                hipMemcpy(gpudata, data, sizeof(hipfftComplex)*nx*batch, hipMemcpyHostToDevice);
                hipDeviceSynchronize();
                cutStopTimer(copy_to_gpu_timer);
                
                cutResetTimer(fft_only_timer);
                cutStartTimer(fft_only_timer);
                hipfftExecC2C(plan,gpudata,fftgpudata,HIPFFT_FORWARD);
                hipDeviceSynchronize();
                cutStopTimer(fft_only_timer);
                
                cutResetTimer(copy_from_gpu_timer);
                cutStartTimer(copy_from_gpu_timer);
                hipMemcpy(result, fftgpudata, sizeof(hipfftComplex)*nx*batch, hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                cutStopTimer(copy_from_gpu_timer);
                
                hipfftDestroy(plan);
                CUDA_SAFE_CALL(hipFree(gpudata));
                CUDA_SAFE_CALL(hipFree(fftgpudata));
            
            
                fprintf(stderr, "%lld\t%lld\t%f\t%f\t%f\t%f\t%f\n",
                    nx, batch, 
                    cutGetTimerValue(complete_fft_timer), cutGetTimerValue(copy_to_gpu_timer), 
                    cutGetTimerValue(fft_only_timer), cutGetTimerValue(copy_from_gpu_timer),
                    cutGetTimerValue(complete_fft_timer)/(nx*batch));
            }
        }
    }
    
    //print fft data
//    for(i=0; i<NX*BATCH; i++)
//    {
//        fprintf(stderr,"%d %f %f\n", i, data[i].x, data[i].y);
//    }
    
    
    return 0;
}
