#include <stdlib.h>
#include <stdio.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#define NX      256
#define BATCH   4


static hipfftHandle plan;
hipfftComplex *gpudata;
hipfftComplex *fftgpudata;


void initializeFFT()
{
    // allocate device memory for the fft
    hipMalloc((void**)&gpudata,sizeof(hipfftComplex)*NX*BATCH);
    hipMalloc((void**)&fftgpudata,sizeof(hipfftComplex)*NX*BATCH);
    
    hipfftPlan1d(&plan,NX,HIPFFT_C2C, BATCH);
}


void callFFT(hipfftComplex *data)
{
    //int i;
    // allocate device memory and copy over data
    hipMemcpy(gpudata, data, sizeof(hipfftComplex)*NX*BATCH, hipMemcpyHostToDevice);
    
    // run the fft
    
    hipfftExecC2C(plan,gpudata,fftgpudata,HIPFFT_FORWARD);
    
    // copy the result back
    hipMemcpy(data, fftgpudata, sizeof(hipfftComplex)*NX*BATCH, hipMemcpyDeviceToHost);
    
    //    for(i=0; i<SAMPLES_PER_CHANNEL*NX*BATCH; i++)
    //    {
    //        fprintf(stderr,"%d %f %f\n", i, data[i].x, data[i].y);
    //    }
    
    
}

void destroyFFT()
{
    hipfftDestroy(plan);
    hipFree(gpudata);
}


int main ()
{
    int i;
    //cufftHandle plan;
    hipfftComplex data[NX*BATCH];
    //cufftComplex *gpudata;
    
    // generate some random data
    for(i=0; i<NX*BATCH; i++)
    {
        data[i].x=1.0f;
        data[i].y=1.0f;
    }
    
    // allocate device memory and copy over data
    initializeFFT();
    
    // run the fft
    callFFT(data);
    
    for(i=0; i<NX*BATCH; i++)
    {
        fprintf(stderr,"%d %f %f\n", i, data[i].x, data[i].y);
    }
    
    destroyFFT();
    
    return 0;
}
